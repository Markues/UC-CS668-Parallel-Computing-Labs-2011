#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <cutil_inline.h>
#include <math.h>
//http://http.developer.nvidia.com/GPUGems3/gpugems3_ch39.html

#if __DEVICE_EMULATION__

bool InitCUDA(void){return true;}

#else

bool InitCUDA(void)
{
	int count = 0;
	int i = 0;

	hipGetDeviceCount(&count);
	if(count == 0) {
		fprintf(stderr, "There is no device.\n");
		return false;
	}

	for(i = 0; i < count; i++) {
		hipDeviceProp_t prop;
		if(hipGetDeviceProperties(&prop, i) == hipSuccess) {
			if(prop.major >= 1) {
				break;
			}
		}
	}
	if(i == count) {
		fprintf(stderr, "There is no device supporting CUDA.\n");
		return false;
	}
	hipSetDevice(i);

	printf("\nCUDA initialized.\n");
	return true;
}
#endif
__device__ static void radixSort(int *A, int *C, int N, int bit)
{
	extern  __shared__  int temp[];
    int idx = threadIdx.x;
	int numFalses = temp[N-1]+1-((int)(A[idx]/pow(2.0,bit)) % 2);
	for(int i=idx;i<N;i++){
		if(1-((int)(A[i]/pow(2.0,bit)) % 2) == 1){
			C[temp[i]] = A[i]; 
		}
		else{
			C[i - temp[i] + numFalses] = A[i];
		}
	}
}
__global__ void bigscan2(int *g_idata,int *C, int *f, int n, int bit)
{
    // Dynamically allocated shared memory for scan kernels
    extern  __shared__  int temp[];
    int thid = threadIdx.x;
	int sectionAdd = 0;
	
	int offset = 1;

	// Cache the computational window in shared memory
	temp[2*thid]   = 1-((int)(g_idata[2*thid]/pow(2.0,bit)) % 2);
	temp[2*thid+1] = 1-((int)(g_idata[2*thid+1]/pow(2.0,bit)) % 2);

	// build the sum in place up the tree
    for (int d = n>>1; d > 0; d >>= 1)
    {
        __syncthreads();

        if (thid < d)      
        {
            int ai = offset*(2*thid+1)-1;
            int bi = offset*(2*thid+2)-1;

            temp[bi] += temp[ai];
        }

        offset *= 2;
    }

    // scan back down the tree

    // clear the last element
    if (thid == 0)
    {
        temp[n - 1] = 0;
    }   

    // traverse down the tree building the scan in place
    for (int d = 1; d < n; d *= 2)
    {
        offset >>= 1;
        __syncthreads();

        if (thid < d)
        {
            int ai = offset*(2*thid+1)-1;
            int bi = offset*(2*thid+2)-1;

            float t   = temp[ai];
            temp[ai]  = temp[bi];
            temp[bi] += t;
        }
    }

	__syncthreads();
	radixSort(g_idata,C,n, bit);
}

__device__ void bigscan(int *g_odata, int *g_idata,int *C, int *f, int n, int bit)
{
    // Dynamically allocated shared memory for scan kernels
    extern  __shared__  int temp[];
    int thidx = 4*threadIdx.x;
	
	for(int thid = thidx;thid < thidx + 4; thid++){
		int offset = 1;
		// Cache the computational window in shared memory
		temp[thid]   = 1-((int)(g_idata[thid]/pow(2.0,bit)) % 2);
		temp[thid+1] = 1-((int)(g_idata[thid+1]/pow(2.0,bit)) % 2);

		// build the sum in place up the tree
		for (int d = n>>1; d > 0; d >>= 1)
		{
			__syncthreads();

			if (thid < d)      
			{
				int ai = offset*(thid+1)-1;
				int bi = offset*(thid+2)-1;

				temp[bi] += temp[ai];
			}

			offset *= 2;
		}

		// scan back down the tree

		// clear the last element
		if (thid == 0)
		{
			temp[n - 1] = 0;
		}   

		// traverse down the tree building the scan in place
		for (int d = 1; d < n; d *= 2)
		{
			offset >>= 1;
			__syncthreads();

			if (thid < d)
			{
				int ai = offset*(thid+1)-1;
				int bi = offset*(thid+2)-1;

				float t   = temp[ai];
				temp[ai]  = temp[bi];
				temp[bi] += t;
			}
		}
		g_odata[thid] = temp[thid];
		g_odata[thid+1] = temp[thid+1];
	}
	

	__syncthreads();
	radixSort(g_idata,C,n, bit);
}
__global__ void scan(int *g_odata, int *g_idata, int *C, int *f, int n, int bit)
{
	extern __shared__ int temp[]; // allocated on invocation
	int thidx = 8*threadIdx.x;
	
	// Load input into shared memory.
	// This is exclusive scan, so shift right by one
	// and set first element to 0
	for(int thid = thidx;thid < thidx + 8; thid++){
		temp[thid] = (thid > 0) ? (((int)(g_idata[thid-1]/pow(2.0,bit)) % 2)) : 0;	
		__syncthreads();
		g_odata[thid] = temp[thid];	
	}
	//bigscan(g_odata,g_idata,C,f,n,bit);
}

int main(int argc, char* argv[])
{
	int  *c, *a, *d_a, *d_c, *d_prefix, *h_prefix, *h_f, *d_f; //host and device arrays
	const int n = 4000; // num elements in array
	size_t size = n * sizeof(int); //size of array
	
	a = (int *)malloc(size);//allocate host array
	c = (int *)malloc(size);//allocate host array
	h_f = (int *)malloc(size);//allocate host array
	h_prefix = (int *)malloc(size);//allocate host array
	hipMalloc((void**) &d_a, size); //allocate device array
	hipMalloc((void**) &d_prefix, size); //allocate device array
	hipMalloc((void**) &d_c, size); //allocate device array
	hipMalloc((void**) &d_f, size); //allocate device array

	//init host array
	for(int i = 0; i < n;i++) a[i] = (int)rand() % 10;//4294967297;	
	printf(" OLD LIST: \n");
	for(int k=0;k < n;++k) printf("%d ",a[k]);

	//copy it to the device 
	
	
	for(int bit = 0; bit <= 0;bit++){ //for each bit
		hipMemcpy(d_a,a,size,hipMemcpyHostToDevice);
		hipMemcpy(d_c,c,size,hipMemcpyHostToDevice);
		hipMemcpy(d_prefix,h_prefix,size,hipMemcpyHostToDevice);
		hipMemcpy(d_f,h_f,size,hipMemcpyHostToDevice);
		scan<<<1,512,size>>>(d_prefix,d_a,d_c,d_f,n,bit);
		//bigscan<<<1,512,size>>>(d_prefix,d_a,d_c,d_f,n,bit);
		hipMemcpy(h_prefix, d_prefix, sizeof(int) * n, hipMemcpyDeviceToHost);
		hipMemcpy(h_f, d_f, sizeof(int) * n, hipMemcpyDeviceToHost);
		hipMemcpy(a, d_c, sizeof(int) * n, hipMemcpyDeviceToHost);		
	}
	printf("\nPREFIX: \n");
	for(int k=0;k < n/2;++k) printf("%d ",h_prefix[k]);
	//printf("\nORDERED LIST: \n");
	
	//for(int k=0;k < n;++k) printf("%d ",a[k]);
	/*bool inOrder = true;
	for(int k=0;k < n;++k) if(a[k] < a[k-1]){inOrder = false;}
	if(inOrder == true){
		printf("\n\n\nIN ORDER!");
	}
	else{
		printf("NOT IN ORDER!");
	}*/
	hipFree(c);
	free(a); 
	return 0;
}
